/*

Naive and Coalesced Transpose of a matrix using CUDA

*/



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 2048
#define BLOCK_DIM 64

void print_matrix(int *mat){
 for(int row=0; row<N; row++) {
  for(int col=0; col<N; col++)
    printf("%d", mat[row*N+col]);
  printf("\n");
 }
};

int compare(int *mat1, int *mat2){
 for(int row=0; row<N; row++) {
  for(int col=0; col<N; col++) {
    if(mat1[row*N+col] != mat2[row*N+col]){
       return 0;
    }
  }
 }
 return 1;
}

__global__ void naive_transpose(int *d_in,int *d_out){
   int ix = blockIdx.x * BLOCK_DIM + threadIdx.x;
   int iy = blockIdx.y * BLOCK_DIM + threadIdx.y;
   if(ix<N && iy<N) {
      d_out[iy+ix*N] = d_in[ix+iy*N];
   }
};

__global__ void transpose_coalesced(int *d_in, int *d_out){
   int ix = blockIdx.x * BLOCK_DIM;
   int iy = blockIdx.y * BLOCK_DIM;
   int ox = blockIdx.y * BLOCK_DIM;
   int oy = blockIdx.x * BLOCK_DIM;
   int x = threadIdx.x, y = threadIdx.y;

   __shared__ int tile[BLOCK_DIM][BLOCK_DIM];

   tile[y][x] = d_in[(ix+x)+(iy+y)*N];

   __syncthreads();

   d_out[(ox+x)+(oy+y)*N] = tile[x][y];
};

int main()
{
 int num_bytes =N*N*sizeof(int);
 int *d_out, *d_in, *h_in, *h_out1,*h_out2;
 float time;
 hipEvent_t start, stop;

 h_in = (int *)malloc(num_bytes);
 h_out1 = (int *)malloc(num_bytes);
 h_out2 = (int *)malloc(num_bytes);

 hipMalloc((void**)&d_in,num_bytes);
 hipMalloc((void**)&d_out,num_bytes);

 hipEventCreate(&start);
 hipEventCreate(&stop);

 for(int row=0; row<N; row++) {
  for(int col=0; col<N; col++)
    h_in[row*N+col] = col;
 }

 hipMemcpy(d_in,h_in,num_bytes,hipMemcpyHostToDevice);

 dim3 blocks(N/BLOCK_DIM,N/BLOCK_DIM);
 dim3 threads(BLOCK_DIM,BLOCK_DIM);

 hipEventRecord(start);
 naive_transpose<<<blocks,threads>>>(d_in,d_out);
 hipEventRecord(stop);

 hipMemcpy(h_out1,d_out,num_bytes,hipMemcpyDeviceToHost);
 hipEventElapsedTime(&time, start, stop);

 printf ("Naive Transpose:\n");
 printf("\n");
 printf ("Time Taken: %f milliseconds\n", time);
 printf("Bandwidth: %f Gb/sec\n", (N*N*4*2*1e-6)/time);
 printf("\n");

 hipEventRecord(start);
 transpose_coalesced<<<blocks,threads>>>(d_in,d_out);
 hipEventRecord(stop);

 hipMemcpy(h_out2,d_out,num_bytes,hipMemcpyDeviceToHost);
 hipEventElapsedTime(&time, start, stop);

 printf ("Coalesced Transpose:\n");
 printf("\n");
 printf ("Time Taken: %f milliseconds\n", time);
 printf("Bandwidth: %f Gb/sec\n", (N*N*4*2*1e-6)/time);
 printf("\n");

 printf("Matrix Comparison: %s\n",compare(h_out1,h_out2) ? "Successful" : "Unsuccessful");
 printf("\n");

 /*
 printf("Matrix Transpose:\n");
 printf("\n");
 print_matrix(h_out1);
 */

 free(h_in);
 free(h_out1);
 free(h_out2);

 hipFree(d_in);
 hipFree(d_out);

 return 0;
}
